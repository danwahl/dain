#include "hip/hip_runtime.h"
#include "kernels.cuh"
#include <stdio.h>

__global__ void add_one_kernel(float* data, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        data[idx] += 1.0f;
    }
}

void add_one_kernel_wrapper(float* input, int size) {
    float* d_array;
    hipError_t err;

    err = hipMalloc(&d_array, size * sizeof(float));
    if (err != hipSuccess) printf("Malloc error: %s\n", hipGetErrorString(err));

    err = hipMemcpy(d_array, input, size * sizeof(float), hipMemcpyHostToDevice);
    if (err != hipSuccess) printf("H2D Memcpy error: %s\n", hipGetErrorString(err));
    
    int block_size = 256;
    int num_blocks = (size + block_size - 1) / block_size;
    add_one_kernel<<<num_blocks, block_size>>>(d_array, size);
    
    err = hipGetLastError();
    if (err != hipSuccess) printf("Kernel error: %s\n", hipGetErrorString(err));

    err = hipMemcpy(input, d_array, size * sizeof(float), hipMemcpyDeviceToHost);
    if (err != hipSuccess) printf("D2H Memcpy error: %s\n", hipGetErrorString(err));

    hipFree(d_array);
}
