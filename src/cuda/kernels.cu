#include "hip/hip_runtime.h"
#include "kernels.cuh"

__global__ void add_kernel(float *a, float *b, float *c, int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size)
    {
        c[idx] = a[idx] + b[idx];
    }
}

__global__ void matmul_kernel(float *a, float *b, float *c, int m, int n, int k)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < m && col < n)
    {
        float sum = 0.0f;
        for (int i = 0; i < k; i++)
        {
            sum += a[row * k + i] * b[i * n + col];
        }
        c[row * n + col] = sum;
    }
}

int dain_add(float *h_a, float *h_b, float *h_c, int size)
{
    float *d_a = nullptr, *d_b = nullptr, *d_c = nullptr;

    hipError_t ret = hipMalloc(&d_a, size * sizeof(float));
    if (ret == hipSuccess)
    {
        ret = hipMalloc(&d_b, size * sizeof(float));
    }
    if (ret == hipSuccess)
    {
        ret = hipMalloc(&d_c, size * sizeof(float));
    }

    if (ret == hipSuccess)
    {
        ret = hipMemcpy(d_a, h_a, size * sizeof(float), hipMemcpyHostToDevice);
    }
    if (ret == hipSuccess)
    {
        ret = hipMemcpy(d_b, h_b, size * sizeof(float), hipMemcpyHostToDevice);
    }

    if (ret == hipSuccess)
    {
        const int block_size = 256;
        const int num_blocks = (size + block_size - 1) / block_size;
        add_kernel<<<num_blocks, block_size>>>(d_a, d_b, d_c, size);
        ret = hipGetLastError();
    }

    if (ret == hipSuccess)
    {
        ret = hipMemcpy(h_c, d_c, size * sizeof(float), hipMemcpyDeviceToHost);
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return ret == hipSuccess ? 0 : 1;
}

int dain_matmul(float *h_a, float *h_b, float *h_c, int m, int n, int k)
{
    float *d_a = nullptr, *d_b = nullptr, *d_c = nullptr;

    hipError_t ret = hipMalloc(&d_a, m * k * sizeof(float));
    if (ret == hipSuccess)
    {
        ret = hipMalloc(&d_b, k * n * sizeof(float));
    }
    if (ret == hipSuccess)
    {
        ret = hipMalloc(&d_c, m * n * sizeof(float));
    }

    if (ret == hipSuccess)
    {
        ret = hipMemcpy(d_a, h_a, m * k * sizeof(float), hipMemcpyHostToDevice);
    }
    if (ret == hipSuccess)
    {
        ret = hipMemcpy(d_b, h_b, k * n * sizeof(float), hipMemcpyHostToDevice);
    }

    if (ret == hipSuccess)
    {
        const dim3 block_dim(16, 16);
        const dim3 grid_dim((n + block_dim.x - 1) / block_dim.x,
                            (m + block_dim.y - 1) / block_dim.y);

        matmul_kernel<<<grid_dim, block_dim>>>(d_a, d_b, d_c, m, n, k);
        ret = hipGetLastError();
    }

    if (ret == hipSuccess)
    {
        ret = hipMemcpy(h_c, d_c, m * n * sizeof(float), hipMemcpyDeviceToHost);
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return ret == hipSuccess ? 0 : 1;
}
