#include "hip/hip_runtime.h"
#include "kernels.cuh"

__global__ void add_one_kernel(float *data, int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size)
    {
        data[idx] += 1.0f;
    }
}

int add_one_kernel_wrapper(float *h_array, int size)
{
    float *d_array = nullptr;
    hipError_t ret = hipMalloc(&d_array, size * sizeof(float));
    if (ret == hipSuccess)
    {
        ret = hipMemcpy(d_array, h_array, size * sizeof(float), hipMemcpyHostToDevice);
    }

    if (ret == hipSuccess)
    {
        const int block_size = 256;
        const int num_blocks = (size + block_size - 1) / block_size;
        add_one_kernel<<<num_blocks, block_size>>>(d_array, size);

        ret = hipGetLastError();
    }

    if (ret == hipSuccess)
    {
        ret = hipMemcpy(h_array, d_array, size * sizeof(float), hipMemcpyDeviceToHost);
    }

    hipFree(d_array);
    return ret == hipSuccess ? 0 : 1;
}
